#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <>//�ڶ���threadIdxʱ����Ҫ�õ���headfiles

__global__  void kernel(float * a) 
{
	a[ threadIdx.x] = 1;
}


int main(int argc, char **argv)
{
	hipSetDevice(0);
	float* aGpu;
	hipMalloc((void**)&aGpu,16* sizeof(float));//�����Դ�ռ�
	float a[16] = {0};//�����ڴ�ռ�
	hipMemcpy(aGpu, a, 16 * sizeof(float), hipMemcpyHostToDevice);//�����ݴ��ڴ濽�����Դ�
	kernel << <1, 16 >> > (aGpu);
	hipMemcpy(a, aGpu, 16 * sizeof(float), hipMemcpyDeviceToHost);
	for (int i = 0; i < 16; i++) {
		std::cout << a[i] << std::endl;
	}
	hipFree(aGpu);//�ͷ��Դ�ռ�
	hipDeviceReset();
    
	int gpuCount = -1;
	hipGetDeviceCount(&gpuCount);
	std::cout << std::endl;
	std::cout << gpuCount << std::endl;


}