#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <>//�ڶ���threadIdxʱ����Ҫ�õ���headfiles

__global__  void kernel(float * a) 
{
	a[ threadIdx.x] = 1;
}


int main(int argc, char **argv)
{
	int gpuCount = -1;
	hipGetDeviceCount(&gpuCount);
	std::cout << "GPU number:" << gpuCount << std::endl;
	if (gpuCount < 0)
		std::cout << "No device to use" << std::endl;

	hipSetDevice(gpuCount-1);
	int deviceID;
	hipGetDevice(&deviceID);
	std::cout << "deviceID :" << deviceID << std::endl;
	float* aGpu;
	hipMalloc((void**)&aGpu,16* sizeof(float));//�����Դ�ռ�
	float a[16] = {0};//�����ڴ�ռ�
	hipMemcpy(aGpu, a, 16 * sizeof(float), hipMemcpyHostToDevice);//�����ݴ��ڴ濽�����Դ�
	kernel << <1, 16 >> > (aGpu);
	hipMemcpy(a, aGpu, 16 * sizeof(float), hipMemcpyDeviceToHost);
	for (int i = 0; i < 16; i++) {
		std::cout << a[i] << std::endl;
	}
	hipFree(aGpu);//�ͷ��Դ�ռ�
	hipDeviceReset();
    
	hipDeviceProp_t pro;//�Կ���һЩ��Ϣ
	hipGetDeviceProperties(&pro,0);
	std::cout<<"maxThreadperBlock:"<<pro.maxThreadsPerBlock <<std::endl;
	std::cout << "maxThreadsDim:" << pro.maxThreadsDim << std::endl;
	std::cout << "maxGridSize:" << pro.maxGridSize << std::endl;
	std::cout << "totalConstMem:" << pro.totalConstMem << std::endl;
	std::cout << "clockRate:" << pro.clockRate << std::endl;
	std::cout << "Intergrated:" << pro.integrated << std::endl;
}

//����7�����裺
//�����Կ��豸
//�����Դ�ռ�
//�������ݴ��ڴ浽�Դ�
//ִ�в��к���
//��������Դ濽���ڴ�
//�ͷ��Դ�ռ�hipFree
//�豸����