#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <>//�ڶ���threadIdxʱ����Ҫ�õ���headfiles
using namespace std;

__global__  void add(int *a , int *b , int * c ,int num)
{
	int i = threadIdx.x;
	if (i < num) {
		c[i] = a[i] + b[i];
	}
}

int main(void) {
	//init data
	const int num = 10;
	int a[num], b[num], c[num];
	int *a_gpu, *b_gpu, *c_gpu;
	for (int i = 0; i < num; i++) {
		a[i] = i;
		b[i] = i * i;
	}
	hipMalloc((void **)&a_gpu, num * sizeof(int));
	hipMalloc((void**)&b_gpu, num * sizeof(int));
	hipMalloc((void**)&c_gpu, num * sizeof(int));
	//copy data to GPU
	hipMemcpy(a_gpu, a, num*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(b_gpu, b, num * sizeof(int), hipMemcpyHostToDevice);
	//do 
	add << <1, num >> > (a_gpu,b_gpu, c_gpu,num);

	hipMemcpy(c, c_gpu, num * sizeof(int), hipMemcpyDeviceToHost);
	//
	for (int i = 0; i < num; i++) {
		cout << a[i] << " + " << b[i] << " = " << c[i] << endl;
	}
	return 0;
}